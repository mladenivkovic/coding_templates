#include <hip/hip_runtime.h>
/* #include <cuda_runtime.h> */
#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

void alloc_array(int** array){
  hipHostMalloc((void**)array, 10 * sizeof(int), hipHostMallocDefault);

  for (int i = 0; i < 10; i++){
    (*array)[i] = i;
  }
  printf("Alloc'd array.\n");
}


void free_array(int** array) {
  hipHostFree((void*)(*array));
  printf("Freed array.\n");
}


#ifdef __cplusplus
}
#endif

