/**
 * Contents of the cuda functions.
 * This should be part of the CUDALIBRARY library.
 */

#ifdef __cplusplus
extern "C" {
#endif

#include <hip/hip_runtime.h>
#include <stdio.h>

/**
 * Allocate an array and fill it up with some data.
 */
void cuda_alloc_array(int** array){

  hipError_t err;
  err = hipHostMalloc((void**)array, 10 * sizeof(int), hipHostMallocDefault);
  if (err != hipSuccess){
    printf("Error allocating array on host. Do you have a GPU?\n");
    fflush(stdout);
    exit(err);
  }

  for (int i = 0; i < 10; i++){
    (*array)[i] = i;
  }
  printf("-- Alloc'd array.\n");
}


void cuda_free_array(int* array) {

  hipError_t err;
  err = hipHostFree((void*)array);

  if (err != hipSuccess){
    printf("Error freeing array on host.");
    fflush(stdout);
    exit(err);
  }

  printf("-- Freed array.\n");
}


#ifdef __cplusplus
}
#endif

