#ifdef __cplusplus
extern "C" {
#endif

#include <hip/hip_runtime.h>
/* #include <cuda_runtime.h> */
#include <stdio.h>

/**
 * Allocate an array and fill it up with some data.
 */
void alloc_array(int** array){

  hipError_t err;
  err = hipHostMalloc((void**)array, 10 * sizeof(int), hipHostMallocDefault);
  if (err != hipSuccess){
    printf("Error allocating array on host. Do you have a GPU?");
    fflush(stdout);
    exit(err);
  }

  for (int i = 0; i < 10; i++){
    (*array)[i] = i;
  }
  printf("Alloc'd array.\n");
}


void free_array(int** array) {

  hipError_t err;
  err = hipHostFree((void*)(*array));

  if (err != hipSuccess){
    printf("Error freeing array on host.");
    fflush(stdout);
    exit(err);
  }

  printf("Freed array.\n");
}


#ifdef __cplusplus
}
#endif

