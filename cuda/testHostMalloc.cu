#ifdef __cplusplus
/* extern "C" { */
#endif


#include <hip/hip_runtime.h>
#include <stdio.h>


int main(void) {

  const int N = 10;

  int *int_p = NULL;
  hipError_t err = hipHostMalloc((void **)&int_p, N * sizeof(int), hipHostMallocDefault);
  if (err != hipSuccess )
    printf("Error allocating memory\n");

  for (int i = 0; i < N; i++)
    int_p[i] = i;

  hipHostFree(int_p);

  printf("Done.\n");
}


#ifdef __cplusplus
/* } */
#endif
